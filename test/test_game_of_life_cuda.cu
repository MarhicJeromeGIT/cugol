#include <gtest/gtest.h>
#include "../cuda/game_of_life_cuda.cuh" // Include your CUDA kernel header

TEST(CUDAKernelTest, StepKernelExecutionTest) {
    GTEST_SKIP();

    const int W = 5;
    const int H = 5;

    int *grid1, *grid2;
    hipMallocManaged(&grid1, H * W * sizeof(int));
    hipMallocManaged(&grid2, H * W * sizeof(int));

    for(int i=0; i < H; i++) {
        for(int j=0; j < W; j++) {
            grid1[i * W + j] = 0;
            grid2[i * W + j] = 0;
        }
    }

    // Initialize a glider:
    grid1[1 * W + 2] = 1;
    grid1[2 * W + 3] = 1;
    grid1[3 * W + 1] = 1;
    grid1[3 * W + 2] = 1;
    grid1[3 * W + 3] = 1;

    // print_grid(W, H, grid1);

    // // Launch kernel and wait for completion
    step<<<1, 1>>>(H, W, grid1, grid2);
    hipDeviceSynchronize();

    // print_grid(W, H, grid2);

    // // Check the result
    // Expected grid values after the kernel execution
    int expectedGrid[H * W] = {
        0, 0, 0, 0, 0, 
        0, 0, 0, 0, 0, 
        0, 1, 0, 1, 0, 
        0, 0, 1, 1, 0, 
        0, 0, 0, 0, 0
    };

    // Check each cell
    for(int i = 0; i < H; ++i) {
        for(int j = 0; j < W; ++j) {
            EXPECT_EQ(grid2[i * W + j], expectedGrid[i * W + j]);
        }
    }

    // // Free device memory
    hipFree(grid1);
    hipFree(grid2);
}

TEST(CUDAKernelTest, SquarePreservationTest) {
    GTEST_SKIP();

    const int W = 5;
    const int H = 5;

    int *grid1, *grid2;
    hipMallocManaged(&grid1, H * W * sizeof(int));
    hipMallocManaged(&grid2, H * W * sizeof(int));

    // Initialize grids to zero
    for(int i = 0; i < H; i++) {
        for(int j = 0; j < W; j++) {
            grid1[i * W + j] = 0;
            grid2[i * W + j] = 0;
        }
    }

    // Initialize a 2x2 square in the center:
    grid1[1 * W + 1] = 1;
    grid1[1 * W + 2] = 1;
    grid1[2 * W + 1] = 1;
    grid1[2 * W + 2] = 1;

    // Launch kernel and wait for completion
    step<<<1, 1>>>(H, W, grid1, grid2);
    hipDeviceSynchronize();

    // Expected grid values after the kernel execution (2x2 square should be preserved)
    int expectedGrid[H * W] = {
        0, 0, 0, 0, 0,
        0, 1, 1, 0, 0,
        0, 1, 1, 0, 0,
        0, 0, 0, 0, 0,
        0, 0, 0, 0, 0
    };

    // Check each cell
    for(int i = 0; i < H; ++i) {
        for(int j = 0; j < W; ++j) {
            EXPECT_EQ(grid2[i * W + j], expectedGrid[i * W + j]);
        }
    }

    // Free device memory
    hipFree(grid1);
    hipFree(grid2);
}

TEST(CUDAKernelTest, SquarePreservationTest2X2) {
    const int W = 8;
    const int H = 8;

    int *grid1, *grid2;
    hipMallocManaged(&grid1, H * W * sizeof(int));
    hipMallocManaged(&grid2, H * W * sizeof(int));

    // Initialize grids to zero
    for(int i = 0; i < H; i++) {
        for(int j = 0; j < W; j++) {
            grid1[i * W + j] = 0;
            grid2[i * W + j] = 0;
        }
    }

    // Initialize a 2x2 square in the center:
    grid1[1 * W + 1] = 1;
    grid1[1 * W + 2] = 1;
    grid1[2 * W + 1] = 1;
    grid1[2 * W + 2] = 1;

    // Launch kernel and wait for completion
    step<<<dim3(4,4), dim3(2,2)>>>(W, H, grid1, grid2);
    hipDeviceSynchronize();

    // Expected grid values after the kernel execution (2x2 square should be preserved)
    int expectedGrid[H * W] = {
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 1, 1, 0, 0, 0, 0, 0,
        0, 1, 1, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0        
    };

    // Check each cell
    for(int i = 0; i < H; ++i) {
        for(int j = 0; j < W; ++j) {
            EXPECT_EQ(grid2[i * W + j], expectedGrid[i * W + j]);
        }
    }

    // Free device memory
    hipFree(grid1);
    hipFree(grid2);
}