# include "../cuda/game_of_life_cuda.cuh"
# include <iostream>
using namespace std;

int main(int argc, char* argv[]) {
    int W = 10; // Width of the grid
    int H = 10; // Height of the grid
    int iterations = 30; // Default iteration count

    // Parse optional command line arguments
    for (int i = 1; i < argc; i++) {
        std::string arg = argv[i];
        if (arg == "-w" && i + 1 < argc) {
            W = std::atoi(argv[++i]);
        } else if (arg == "-h" && i + 1 < argc) {
            H = std::atoi(argv[++i]);
        } else if (arg == "-iteration" && i + 1 < argc) {
            iterations = std::atoi(argv[++i]);
        }
    }

    std::cout << "Width: " << W << ", Height: " << H << ", Iterations: " << iterations << std::endl;

    // Initialize an empty grid:
    int *grid1, *grid2;
    hipMallocManaged(&grid1, H * W * sizeof(int));
    hipMallocManaged(&grid2, H * W * sizeof(int));
    // For now let's say that the edge cells are always empty (dead)

    for(int i=0; i < H; i++) {
        for(int j=0; j < W; j++) {
            grid1[i * W + j] = 0;
            grid2[i * W + j] = 0;
        }
    }

    // Initialize a glider:
    grid1[1 * W + 2] = 1;
    grid1[2 * W + 3] = 1;
    grid1[3 * W + 1] = 1;
    grid1[3 * W + 2] = 1;
    grid1[3 * W + 3] = 1;

    // print(grid1);
    int smCount; // We can set it to the number of SMs
    hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, 0);
    cout << "SM count : " << smCount << endl;

    for(int i=0; i < iterations; i++) {
        step<<<smCount*4, 256>>>(H, W, grid1, grid2);
        hipDeviceSynchronize();

        hipError_t cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(1);
        }

        swapGrids<<<smCount*4, 256>>>(H, W, grid1, grid2);
        hipDeviceSynchronize();
        // print_grid(W, H, grid1);
    }

    // print_grid(W, H, grid1);

    // Free the allocated memory
    hipFree(grid1);
    hipFree(grid2);

    return 0;
}